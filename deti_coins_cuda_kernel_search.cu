#include "hip/hip_runtime.h"
//
// Gabriel Negri, Marco Magalhães, December 2024
//
// Arquiteturas de Alto Desempenho 2024/2025
//
// DETI coins search CUDA kernel code
//
// deti_coins_cuda_kernel_search() --- each thread tries 1 potential DETI coins
//

//
// needed stuff
//

typedef unsigned int u32_t;
typedef unsigned int u08_t;

#include "md5.h"

//
// the nvcc compiler stores x[] and state[] in registers (constant indices!)
//
// global thread number: n = threadIdx.x + blockDim.x * blockIdx.x
// global warp number: n >> 5
// warp thread number: n & 31
//

extern "C" __global__ __launch_bounds__(128,1) void deti_coins_cuda_kernel_search(u32_t *deti_coins, u32_t v1, u32_t v2)
{
  u32_t n,a,b,c,d,coin[13],hash[4],state[4],x[16];

  //
  // get the global thread number
  //
  n = (u32_t)threadIdx.x + (u32_t)blockDim.x * (u32_t)blockIdx.x;
  //
  // create DETI coin for thread n
  //
  coin[0] = ('I' << 24) | ('T' << 16) | ('E' << 8) | 'D';
  coin[1] = ('i' << 24) | ('o' << 16) | ('c' << 8) | ' ';
  coin[2] = (' ' << 24) | (' ' << 16) | (' ' << 8) | 'n';
  coin[3] = (' ' << 24) | (' ' << 16) | (' ' << 8) | ' ';
  coin[4] = v1;
  coin[5] = v2;
  coin[6] = (' ' << 24) | (' ' << 16) | (' ' << 8) | ' ';
  coin[7] = (' ' << 24) | (' ' << 16) | (' ' << 8) | ' ';
  coin[8] = (' ' << 24) | (' ' << 16) | (' ' << 8) | ' ';
  coin[9] = (' ' << 24) | (' ' << 16) | (' ' << 8) | ' ';
  coin[10] = (' ' << 24) | (' ' << 16) | (' ' << 8) | ' ';
  coin[11] = (' ' << 24) | (' ' << 16) | (' ' << 8) | ' ';
  coin[12] = ('\n' << 24) | (' ' << 16) | (' ' << 8) | ' ';

  coin[3] += (n % 64) << 0;
  n /= 64;
  coin[3] += (n % 64) << 8;
  n /= 64;
  coin[3] += (n % 64) << 16;
  n /= 64;
  coin[3] += (n % 64) << 24;

  for (n = 0; n < 64; n++) {
    //
    // compute MD5 hash
    //
    # define C(c)         (c)
    # define ROTATE(x,n)  (((x) << (n)) | ((x) >> (32 - (n))))
    # define DATA(idx)    coin[idx]
    # define HASH(idx)    hash[idx]
    # define STATE(idx)   state[idx]
    # define X(idx)       x[idx]
      CUSTOM_MD5_CODE();
    # undef C
    # undef ROTATE
    # undef DATA
    # undef HASH
    # undef STATE
    # undef X

    //
    // validate potential DETI coin
    //
    if (hash[3u] == 0u) {
      a = atomicAdd(deti_coins, 13u);
      if (a <= 1024u - 13u) {
        deti_coins[a] = coin[0u];
        deti_coins[a+1u] = coin[1u];
        deti_coins[a+2u] = coin[2u];
        deti_coins[a+3u] = coin[3u];
        deti_coins[a+4u] = coin[4u];
        deti_coins[a+5u] = coin[5u];
        deti_coins[a+6u] = coin[6u];
        deti_coins[a+7u] = coin[7u];
        deti_coins[a+8u] = coin[8u];
        deti_coins[a+9u] = coin[9u];
        deti_coins[a+10u] = coin[10u];
        deti_coins[a+11u] = coin[11u];
        deti_coins[a+12u] = coin[12u];
      }  
    }
    coin[12u] += (1 << 16);
  }
}